//xfail:NOT_ALL_VERIFIED
//--blockDim=32 --gridDim=2

#include <hip/hip_runtime.h>

__global__ void race (int* A)
{
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  int idx = blockDim.x * bid + tid;

  int temp = A[idx + 1];
  A[idx] = temp;
}
